#include "hip/hip_runtime.h"

#include <iostream>

__global__ void vectorAdd(int *a, int *b, int *c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  int a[] = {1, 2, 3};
  int b[] = {4, 5, 6};
  int c[sizeof(a) / sizeof(int)] = {0};

  int *cudaA = 0;
  int *cudaB = 0;
  int *cudaC = 0;

  hipMalloc(&cudaA, sizeof(a));
  hipMalloc(&cudaB, sizeof(b));
  hipMalloc(&cudaC, sizeof(c));
  hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
  hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

  vectorAdd<<<1, sizeof(a) / sizeof(int)>>>(cudaA, cudaB, cudaC);

  hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

  for (int i = 0; i < sizeof(a) / sizeof(int); i++) {
    std::cout << c[i] << " ";
  }
  std::cout << std::endl;

  hipFree(cudaA);
  hipFree(cudaB);
  hipFree(cudaC);

  return 0;
}
